
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define BLOCK_SIZE 256

__global__ void reduce(float *input, float *output, int n) {
    extern __shared__ float sharedData[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Chargement des données de l'entrée globale à la mémoire partagée
    if (i < n) {
        sharedData[tid] = input[i] + (i + blockDim.x < n ? input[i + blockDim.x] : 0);
    } else {
        sharedData[tid] = 0;
    }
    __syncthreads();

    // Réduction en parallèle dans la mémoire partagée
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }

    // Le premier thread écrit le résultat dans la mémoire globale
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

// Fonction d'appel pour la réduction
float reduceVector(float *h_input, int n) {
    float *d_input, *d_output;
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, numBlocks * sizeof(float));

    hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);

    // Premier passage de réduction
    reduce<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(d_input, d_output, n);

    // Si plus d'un bloc, il faut encore réduire les résultats intermédiaires
    while (numBlocks > 1) {
        n = numBlocks;
        numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        reduce<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(d_output, d_output, n);
    }

    float result;
    hipMemcpy(&result, d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return result;
}

void
initializevector (float *f, int n, int i)
{
  for (int idx = 0; idx < n; ++idx)
    {
      f[idx]
          = (float)drand48 () * i /1e1 ; // warning, requires posix standard.
    }
}

double
doubledotproduct (double *pv1, double *pv2, int n)
{
  double res = 0.0;
  for (int i = 0; i < n; i++)
    res += pv1[i];

  return res;
}
float
floatdotproduct (float *pv1, float *pv2, int n)
{

  float res = 0.0;
  for (int i = 0; i < n; i++)
    res += pv1[i];

  return res;
}
void
transfertodouble (double *pv, float *fv, int n)
{
  for (int idx = 0; idx < n; ++idx)
    {
      pv[idx] = (double)fv[idx];
    }
}
void
transfertocuda (float *pv, float *fv, int n)
{
  for (int idx = 0; idx < n; ++idx)
    {
      pv[idx] = fv[idx];
    }
}

int
main (int argc, char *argv[])
{

float *f1, *f2;
double *d1, *d2;
  FILE *file;
  file = fopen ("result.dat", "w");
  fprintf (file, "n;float; posit32; double; P32-Double; double-float\n");
  for (int i = -100000; i < 100000; i += 1)
    {
      int n = 1000;
      f1 = (float*)malloc (sizeof (float) * n);
      f2 = (float*)malloc (sizeof (float) * n);

      d1 = (double *)malloc (sizeof (double) * n);
      d2 = (double *)malloc (sizeof (double) * n);

      initializevector (f1, n, i);
      initializevector (f2, n, i);

      transfertodouble (d1, f1, n);
      transfertodouble (d2, f2, n);

      float f_res = floatdotproduct (f1, f2, n);
      float cuda_res = reduceVector(f1, n);
      double d_res = doubledotproduct (d1, d2, n);

      fprintf (file, "%d; %24.23lf; %24.23lf; %24.23lf; %e; %e \n", i,
               (double)f_res, (double)cuda_res, d_res,
               ((double)cuda_res - d_res) / d_res, (f_res - d_res) / d_res);
    }
  return 0;
}
