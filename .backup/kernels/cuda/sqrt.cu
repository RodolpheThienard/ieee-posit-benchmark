#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__global__ void
square_root_newton_raphson (float *input, float *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float approx = input[tid] * .5;
  if (input[tid] < 0)
    {
      return;
    }
  if (input[tid] > 0)
    {
      do
        {
          approx = (approx + (input[tid] / approx)) / 2;
        }
      while (((approx * approx) - input[tid]) > 1e-8);
    }
  output[tid] = approx;
}

__global__ void
sqrt_libmath (float *input, float *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
    output[tid] = sqrt (input[tid]);
}
