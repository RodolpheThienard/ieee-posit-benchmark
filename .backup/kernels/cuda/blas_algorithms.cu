
#include <hip/hip_runtime.h>
__global__ void
sgemm (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
vector_add (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
      C[i+j] = A[i+j] * B[i+j];
}

__global__ void
vector_dotprod (float *A, float *B, float *C, int N)
{
  extern __shared__ float S[];
  int tid = threadIdx.x ;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    S[tid] = A[i];
    __syncthreads();
    // Reduce in shared memory
    for (int t = blockDim.x/2; t > 0; t>>=1) {
      if (tid < t) {
        S[tid] += S[tid + t];
      }
      __syncthreads();
    }
    if (tid == 0) C[blockIdx.x] = S[0];
  }
}

__global__ void
vector_div (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
      C[i+j] = A[i+j] / B[i+j];
}

__global__ void
vector_sqrt (float *A, float *B, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
      B[i+j] = sqrt(A[i+j]);
}
