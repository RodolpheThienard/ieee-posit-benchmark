#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__device__ float 
factorial(int n) {
    float result = 1.0;
    int i;
    for (i = 2; i <= n; i++) {
        result *= i;
    }
    return result;
}

__global__ void
sinus_maclaurin (float *input, float *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int  j;
      float result = 0.0;
      for (j = 0; j < 10; j++)
        {
          int sign = (j % 2 == 0) ? 1 : -1;
          int exponent = 2 * j + 1;
          float term
              = sign * pow (input[tid], exponent) / factorial(exponent);
          result += term;
        }
      output[tid] = result;
    
}

__global__ void
sinus_libmath (float *input, float *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
      output[tid] = sin (input[tid]);
}
