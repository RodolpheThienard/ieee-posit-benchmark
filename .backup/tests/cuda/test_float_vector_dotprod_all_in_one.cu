
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void
vector_dotprod (float *A, float *B, int N)
{
  extern __shared__ float S[];
  int tid = threadIdx.x ;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    S[tid] = A[i];
    __syncthreads();
    // Reduce in shared memory
    for (int t = blockDim.x/2; t > 0; t>>=1) {
      if (tid < t) {
        S[tid] += S[tid + t];
      }
      __syncthreads();
    }
    if (tid == 0) B[blockIdx.x] = S[0];
  }
}

__global__ void reduction(float *g_idata, float* g_odata, int n){
  extern __shared__ float sdata[];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();
  // do reduction in shared mem
  for(unsigned int s=1; s < blockDim.x; s *= 2) {
    if (tid % (2*s) == 0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

void
initializevector (float *f, int n, int i)
{
  for (int idx = 0; idx < n; ++idx)
    {
      f[idx]
          = (float)drand48 () * i / 100; // warning, requires posix standard.
    }
}

double
doubledotproduct (double *pv1, double *pv2, int n)
{
  double res = 0.0;
  for (int i = 0; i < n; i++)
    res += pv1[i];

  return res;
}
float
floatdotproduct (float *pv1, float *pv2, int n)
{

  float res = 0.0;
  for (int i = 0; i < n; i++)
    res += pv1[i];

  return res;
}
void
transfertodouble (double *pv, float *fv, int n)
{
  for (int idx = 0; idx < n; ++idx)
    {
      pv[idx] = (double)fv[idx];
    }
}
void
transfertocuda (float *pv, float *fv, int n)
{
  for (int idx = 0; idx < n; ++idx)
    {
      pv[idx] = fv[idx];
    }
}

int
main (int argc, char *argv[])
{

float *f1, *f2, *c1, *d_c1, *d_c2;
double *d1, *d2;
  FILE *file;
  file = fopen ("result.dat", "w");
  fprintf (file, "n;float; posit32; double; P32-Double; double-float\n");
  for (int i = -100000; i < 100000; i += 1)
    {
      int n = 1000;
      f1 = (float*)malloc (sizeof (float) * n);
      f2 = (float*)malloc (sizeof (float) * n);

      c1 = (float *)malloc (sizeof (float) * n);

      hipMalloc (&d_c1, n * sizeof (float));
      hipMalloc (&d_c2, n * sizeof (float));

      d1 = (double *)malloc (sizeof (double) * n);
      d2 = (double *)malloc (sizeof (double) * n);

      initializevector (f1, n, i);
      initializevector (f2, n, i);

      transfertodouble (d1, f1, n);
      transfertodouble (d2, f2, n);


      hipMemcpy (d_c1, f1, n * sizeof (float),
                  hipMemcpyHostToDevice);

      reduction<<<1255/n, n/2>>>(d_c1, d_c2, n);

      hipMemcpy (c1, d_c2, n * sizeof (float),
                  hipMemcpyDeviceToHost);

      float f_res = floatdotproduct (f1, f2, n);
      float cuda_res = c1[0];
      double d_res = doubledotproduct (d1, d2, n);

      fprintf (file, "%d; %24.23lf; %24.23lf; %24.23lf; %e; %e \n", i,
               (double)f_res, (double)cuda_res, d_res,
               ((double)cuda_res - d_res) / d_res, (f_res - d_res) / d_res);
    }
  return 0;
}
