
#include <hip/hip_runtime.h>
__global__ void
sgemm (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
vector_add (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
      C[i+j] = A[i+j] * B[i+j];
}

__global__ void
vector_div (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
      C[i+j] = A[i+j] / B[i+j];
}

__global__ void
vector_sqrt (float *A, float *B, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
      B[i+j] = sqrt(A[i+j]);
}
