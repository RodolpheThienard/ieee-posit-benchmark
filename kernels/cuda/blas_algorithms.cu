
#include <hip/hip_runtime.h>
void
ieee_32bits_gemm (float * a, float * b, float * c,
                  int n)
{
  for (int i = 0; i < n; i++)
    {
      for (int k = 0; k < n; k++)
        {
          const float _a_ = a[i * n + k];

          for (int j = 0; j < n; j++)
            c[i * n + j] += _a_ * b[k * n + j];
        }
    }
}

void
ieee_64bits_gemm (double * a, double * b, double * c,
                  int n)
{
  for (int i = 0; i < n; i++)
    {
      for (int k = 0; k < n; k++)
        {
          const double _a_ = a[i * n + k];

          for (int j = 0; j < n; j++)
            c[i * n + j] += _a_ * b[k * n + j];
        }
    }
}

__global__ void 
sgemm (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}
__global__ void
dgemm (double *A, double *B, double *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
ieee_64bits_gemm_jik (double *A, double *B, double *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
ieee_64bits_gemm_ijk (double *A, double *B, double *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
ieee_64bits_gemm_bloc (double *A, double *B, double *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ double a[32][32];
  __shared__ double b[32][32];

  double sum = 0;
  for (int k = 0; k < N; k += 32)
    {
      a[threadIdx.y][threadIdx.x] = A[j * N + k + threadIdx.x];
      b[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + i];

      __syncthreads ();

      for (int l = 0; l < 32; l++)
        sum += a[threadIdx.y][l] * b[l][threadIdx.x];

      __syncthreads ();
    }
  C[j * N + i] = sum;
}


