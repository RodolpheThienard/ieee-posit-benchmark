
#include <hip/hip_runtime.h>
__global__ void 
sgemm (float *A, float *B, float *C, int N)
{
  int j= blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}
