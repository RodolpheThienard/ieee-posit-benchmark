#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__device__ double 
factorial(int n) {
    double result = 1.0;
    int i;
    for (i = 2; i <= n; i++) {
        result *= i;
    }
    return result;
}

__global__ void
sinus_maclaurin (double *vector, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int  j;
      double result = 0.0;
      for (j = 0; j < 10; j++)
        {
          int sign = (j % 2 == 0) ? 1 : -1;
          int exponent = 2 * j + 1;
          double term
              = sign * pow (vector[tid], exponent) / factorial(exponent);
          result += term;
        }
      vector[tid] = result;
    
}

__global__ void
sinus_libmath (double *vector, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
      vector[tid] = sin (vector[tid]);
}
