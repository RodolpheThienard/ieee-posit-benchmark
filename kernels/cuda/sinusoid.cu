#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__device__ double 
factorial(int n) {
    double result = 1.0;
    int i;
    for (i = 2; i <= n; i++) {
        result *= i;
    }
    return result;
}

__global__ void
sinus_maclaurin (double *input, double *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int  j;
      double result = 0.0;
      for (j = 0; j < 10; j++)
        {
          int sign = (j % 2 == 0) ? 1 : -1;
          int exponent = 2 * j + 1;
          double term
              = sign * pow (input[tid], exponent) / factorial(exponent);
          result += term;
        }
      output[tid] = result;
    
}

__global__ void
sinus_libmath (double *input, double *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
      output[tid] = sin (input[tid]);
}
