
#include <hip/hip_runtime.h>

__global__ void
log_libmath (double *x, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      x[tid] = log (x[tid]);
    }
}

__global__ void
logarithm_taylor (double *x, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      int iter = 100;
      double result = 0.0;
      double term = x[tid] - 1.0;

      for (int i = 1; i <= iter; i++)
        {
          if (i % 2 == 0)
            {
              result -= term / i;
            }
          else
            {
              result += term / i;
            }
          term *= (x[tid] - 1.0);
        }

      x[tid] = result;
    }
}
