
#include <hip/hip_runtime.h>

__global__ void
log_libmath (float *input, float *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      output[tid] = log (input[tid]);
    }
}

__global__ void
logarithm_taylor (float *input, float *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      int iter = 100;
      float result = 0.0;
      float term = input[tid] - 1.0;

      for (int i = 1; i <= iter; i++)
        {
          if (i % 2 == 0)
            {
              result -= term / i;
            }
          else
            {
              result += term / i;
            }
          term *= (input[tid] - 1.0);
        }

      output[tid] = result;
    }
}
