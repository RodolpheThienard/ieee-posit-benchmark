
#include <hip/hip_runtime.h>

__global__ void
log_libmath (double *input, double *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      output[tid] = log (input[tid]);
    }
}

__global__ void
logarithm_taylor (double *input, double *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      int iter = 100;
      double result = 0.0;
      double term = input[tid] - 1.0;

      for (int i = 1; i <= iter; i++)
        {
          if (i % 2 == 0)
            {
              result -= term / i;
            }
          else
            {
              result += term / i;
            }
          term *= (input[tid] - 1.0);
        }

      output[tid] = result;
    }
}
