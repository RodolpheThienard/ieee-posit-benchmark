#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__global__ void
conversion_double_float (double *a, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float tmp;
      tmp = (float)a[tid];
      a[tid] = (double)tmp;
}

__global__ void
useless_function (double *a, int i)
{
}

