#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__global__ void
square_root_newton_raphson (double *vector, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double approx = vector[tid] * .5;
  if (vector[tid] < 0)
    {
      return;
    }
  if (vector[tid] > 0)
    {
      do
        {
          approx = (approx + (vector[tid] / approx)) / 2;
        }
      while (((approx * approx) - vector[tid]) > 1e-8);
    }
  vector[tid] = approx;
}

__global__ void
sqrt_libmath (double *x, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
    x[tid] = sqrt (x[tid]);
}
