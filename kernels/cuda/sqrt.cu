#include "hip/hip_runtime.h"
#include "../../include/utils.h"

__global__ void
square_root_newton_raphson (double *input, double *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double approx = input[tid] * .5;
  if (input[tid] < 0)
    {
      return;
    }
  if (input[tid] > 0)
    {
      do
        {
          approx = (approx + (input[tid] / approx)) / 2;
        }
      while (((approx * approx) - input[tid]) > 1e-8);
    }
  output[tid] = approx;
}

__global__ void
sqrt_libmath (double *input, double *output, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
    output[tid] = sqrt (input[tid]);
}
