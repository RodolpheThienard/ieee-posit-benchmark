
#include <hip/hip_runtime.h>
void
inve_matrix_gauss_jordan_double (double * mat, double * inv, int n)
{
  double *temp = (double*)malloc (n * n * sizeof (double));
  for (int i = 0; i < n * n; i++)
    {
      temp[i] = mat[i];
    }

  // init identity matrix
  for (int i = 0; i < n; i++)
    {
      for (int j = 0; j < n; j++)
        {
          if (i == j)
            {
              inv[i * n + j] = 1.0;
            }
          else
            {
              inv[i * n + j] = 0.0;
            }
        }
    }

  // Gauss-Jordan elimination
  for (int i = 0; i < n; i++)
    {
      // get pivot
      double pivot = temp[i * n + i];

      // Dividing by pivot
      for (int j = 0; j < n; j++)
        {
          temp[i * n + j] /= pivot;
          inv[i * n + j] /= pivot;
        }

      // remove other elements
      for (int k = 0; k < n; k++)
        {
          if (k != i)
            {
              double coeff = temp[k * n + i];
              for (int j = 0; j < n; j++)
                {
                  temp[k * n + j] -= coeff * temp[i * n + j];
                  inv[k * n + j] -= coeff * inv[i * n + j];
                }
            }
        }
    }

  // free memory
  free (temp);
}

__global__ void
inve_matrix_gauss_jordan (float *mat, float *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          float tmp = mat[i * n + i];
          ;
          // get pivot
          float pivot = tmp;

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              float coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}

__global__ void
inve_matrix_gauss_jordan_2 (float *a, float *b, int size)
{
#define element(_x, _y) (*(sdata + ((_y) * (size + 1) + (_x))))
  unsigned int xx, yy, rr;

  // With a limit of 512 threads per block, and only one block, this results
  // in a maximum of a matrix size 22, which requires (22 + 1) x 22 values
  __shared__ float sdata[(22 + 1) * 22];

  xx = threadIdx.x;
  rr = threadIdx.y;

  int tid = rr * (size + 1) + xx;

  // The matrix will be modified in place, so first make a copy of matrix a
  sdata[tid] = a[tid];

  for (yy = 0; yy < size; yy++)
    {

      __syncthreads ();

      // Make the pivot be 1
      element (xx, yy) /= element (yy, yy);

      __syncthreads ();

      // Make all other values in the pivot column be zero
      if (rr != yy)
        element (xx, rr) -= element (yy, rr) * element (xx, yy);
    }

  b[tid] = sdata[tid];
#undef element
}
