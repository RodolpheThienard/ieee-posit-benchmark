
#include <hip/hip_runtime.h>
#include <cstdio>
void
inve_matrix_gauss_jordan_double (double * mat, double * inv, int n)
{
  double *temp = (double*)malloc (n * n * sizeof (double));
  for (int i = 0; i < n * n; i++)
    {
      temp[i] = mat[i];
    }

  // init identity matrix
  for (int i = 0; i < n; i++)
    {
      for (int j = 0; j < n; j++)
        {
          if (i == j)
            {
              inv[i * n + j] = 1.0;
            }
          else
            {
              inv[i * n + j] = 0.0;
            }
        }
    }

  // Gauss-Jordan elimination
  for (int i = 0; i < n; i++)
    {
      // get pivot
      double pivot = temp[i * n + i];

      // Dividing by pivot
      for (int j = 0; j < n; j++)
        {
          temp[i * n + j] /= pivot;
          inv[i * n + j] /= pivot;
        }

      // remove other elements
      for (int k = 0; k < n; k++)
        {
          if (k != i)
            {
              double coeff = temp[k * n + i];
              for (int j = 0; j < n; j++)
                {
                  temp[k * n + j] -= coeff * temp[i * n + j];
                  inv[k * n + j] -= coeff * inv[i * n + j];
                }
            }
        }
    }

  // free memory
  free (temp);
}

__global__ void
inve_matrix_gauss_jordan (float *mat, float *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          float tmp = mat[i * n + i];
          ;
          // get pivot
          float pivot = tmp;

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              float coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}

__global__ void
inve_matrix_gauss_jordan_2 (float *a, float *b, int size, int i)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < size && y < size)
    {
      float pivot = a[i * size + i];
      if (x == i && x != y)
        {
          b[x * size + y] /= pivot;
          a[x * size + y] /= pivot;
        }

      if (x == y && x == i)
        {
          b[x * size + y] /= pivot;
          a[x * size + y] /= pivot;
        }

      if (x != i)
        {
          float multiplier = a[x * size + i];
          b[x * size + y] -= b[i * size + y] * multiplier;
          if (y != i)
            {
              a[x * size + y] -= a[i * size + y] * multiplier;
            }
        }

      if (x != i && y == i)
        {
          a[x * size + y] = 0;
        }
    }
}
