
#include <hip/hip_runtime.h>
__global__ void
inve_matrix_gauss_jordan_2 (float *mat, float *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          float tmp = mat[i * n + i];
          ;
          // get pivot
          float pivot = tmp;

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              float coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}

__global__ void
inve_matrix_gauss_jordan_cuda(float *a, float *b, int size, int i)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < size && y < size)
    {
      float pivot = a[i * size + i];
      if (x == i && x != y)
        {
          b[x * size + y] /= pivot;
          a[x * size + y] /= pivot;
        }

      if (x == y && x == i)
        {
          b[x * size + y] /= pivot;
          a[x * size + y] /= pivot;
        }

      if (x != i)
        {
          float multiplier = a[x * size + i];
          b[x * size + y] -= b[i * size + y] * multiplier;
          if (y != i)
            {
              a[x * size + y] -= a[i * size + y] * multiplier;
            }
        }

      if (x != i && y == i)
        {
          a[x * size + y] = 0;
        }
    }
}
