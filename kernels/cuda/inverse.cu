
#include <hip/hip_runtime.h>
__global__ void
inve_matrix_gauss_jordan_2 (double *mat, double *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          double tmp = mat[i * n + i];
          ;
          // get pivot
          double pivot = tmp;

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              double coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}

__global__ void
inve_matrix_gauss_jordan (double *a, double *b, int size)
{
#define element(_x, _y) (*(sdata + ((_y) * (size + 1) + (_x))))
  unsigned int xx, yy, rr;

  // With a limit of 512 threads per block, and only one block, this results
  // in a maximum of a matrix size 22, which requires (22 + 1) x 22 values
  __shared__ float sdata[(22 + 1) * 22];

  xx = threadIdx.x;
  rr = threadIdx.y;

  int tid = rr * (size + 1) + xx;

  // The matrix will be modified in place, so first make a copy of matrix a
  sdata[tid] = a[tid];

  for (yy = 0; yy < size; yy++)
    {

      __syncthreads ();

      // Make the pivot be 1
      element (xx, yy) /= element (yy, yy);

      __syncthreads ();

      // Make all other values in the pivot column be zero
      if (rr != yy)
        element (xx, rr) -= element (yy, rr) * element (xx, yy);
    }

  b[tid] = sdata[tid];
#undef element
}
