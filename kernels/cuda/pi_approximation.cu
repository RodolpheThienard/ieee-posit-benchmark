#include "hip/hip_runtime.h"
extern "C"
{
  #include "../../include/utils.h"
}
#include <hiprand/hiprand_kernel.h>
#define TOT_DART 1e7
#define PI 3.1415926535
#define is_inside(x, y) (x * x + y * y) <= 1 ? 1 : 0

__global__ void
pi_approximation (float *input, float *output, int n)
{

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j+i > n)return;
  int inside = 0;
  float x, y;
  hiprandState state;
  hiprand_init (j, i, 0, &state);
  for (int k = 0; k < TOT_DART; k++)
    {
      x = (float)hiprand_uniform (&state);
      y = (float)hiprand_uniform (&state);
      inside += is_inside (x, y);
    }
      output[i+j] = (4. * inside / TOT_DART);
}
