#include <cstddef>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define max(a, b) ((a) < (b) ? (b) : (a))
#define gettime(t) clock_gettime (CLOCK_MONOTONIC_RAW, t)
#define get_sub_seconde(t) (1e-9 * (float)t.tv_nsec)

__global__ void reduction(int *output, int *input, int n) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = 0; 
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 32; s /= 2) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[0] = sdata[0];
    }
}


__device__ void
warpReduce4 (volatile int *sdata, int tid)
{
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

__global__ void
reduce (int *g_odata, int *g_idata, int n)
{
  extern __shared__ int sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
  __syncthreads ();

  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
      if (tid < s)
        sdata[tid] += sdata[tid + s];
      __syncthreads ();
    }
  if (tid < 32)
    warpReduce4 (sdata, tid);

  if (tid == 0)
    g_odata[blockIdx.x] = sdata[0];

}

int
ma (int argc, char *argv[])
{
  hipError_t err = hipGetLastError ();
  int N = 2 << 10;
  int *h_input, *d_input, *h_output, *d_output;

  h_input = (int *)malloc (sizeof (int) * N);
  h_output = (int *)malloc (sizeof (int) * N);

  // init host values
  for (int i = 0; i < N; i++)
    {
      h_input[i] = 1;
      h_output[i] = 0;
    }

  hipMalloc (&d_input, sizeof (int) * N);
  hipMalloc (&d_output, sizeof (int) * N);

  hipMemcpy (d_input, h_input, N * sizeof (int), hipMemcpyHostToDevice);
  hipMemcpy (d_output, h_output, N * sizeof (int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock (32);
  dim3 numBlocks ((N+31)/32);

  reduction<<<numBlocks, threadsPerBlock>>> (d_output, d_input, N);

  hipMemcpy (h_output, d_output, N * sizeof (int), hipMemcpyDeviceToHost);

  err = hipGetLastError ();
  printf ("ERROR : %s\n", hipGetErrorString (err));

  printf ("Data gathered : %d; %d\n", h_input[0], h_output[0]);

  hipFree (d_input);
  hipFree (d_output);
  free (h_input);
  free (h_output);
  return 0;
}
