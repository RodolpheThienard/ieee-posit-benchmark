
#include <hip/hip_runtime.h>
__global__ void
ieee_64bits_gemm_jik (double *A, double *B, double *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
ieee_64bits_gemm_ijk (double *A, double *B, double *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
ieee_64bits_gemm_bloc (double *A, double *B, double *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ double a[32][32];
  __shared__ double b[32][32];

  double sum = 0;
  for (int k = 0; k < N; k += 32)
    {
      a[threadIdx.y][threadIdx.x] = A[j * N + k + threadIdx.x];
      b[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + i];

      __syncthreads ();

      for (int l = 0; l < 32; l++)
        sum += a[threadIdx.y][l] * b[l][threadIdx.x];

      __syncthreads ();
    }
  C[j * N + i] = sum;
}

__global__ void
inve_matrix_gauss_jordan (double *mat, double *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          // get pivot
          double pivot = mat[i * n + i];

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              double coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}
