
#include <hip/hip_runtime.h>
__global__ void
cuda_64bits_gemm_jik (double *A, double *B, double *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void
cuda_64bits_gemm_ijk (double *A, double *B, double *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void 
cuda_64bits_gemm_bloc(double *A, double *B, double *C, int N)
{
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     int j = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ double a[32][32];
    __shared__ double b[32][32];

    double sum = 0;
    for (int k = 0; k < N; k += 32) {
        a[threadIdx.y][threadIdx.x] = A[j * N + k + threadIdx.x];
        b[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + i];

        __syncthreads();

        for (int l = 0; l < 32; l++)
            sum += a[threadIdx.y][l] * b[l][threadIdx.x];

        __syncthreads();
    }
    C[j * N + i] = sum;
}
