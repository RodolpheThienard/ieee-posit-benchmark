#include <cstddef>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define max(a, b) ((a) < (b) ? (b) : (a))
#define gettime(t) clock_gettime (CLOCK_MONOTONIC_RAW, t)
#define get_sub_seconde(t) (1e-9 * (float)t.tv_nsec)

/* Masses : Same for all particles
   Acceleration : differ
   Velocity : differ */
struct system {
  double *x, *y;
  double *acceleration, *velocity, masses;
};

/* Kernel of nbody function */
__global__ void
nbody_update (struct system system,  int N)
{

  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      {
        system.x[i * N + j] += 1;
        system.y[i * N + j] += 1;
      }
  return;
}

// Initialisation with linear congruential algorithm and 48-bit integer arithmetic
void
init (struct system system, int N)
{
  for (int i = 0; i < N; i++)
    {
      system.x[i] = drand48 ();
      system.y[i] = drand48 ();
    }
}


int
ma (int argc, char *argv[])
{
  int N = 42;
  double *d_x, *d_y, *d_acceleration, *d_velocity;
  double *h_x, *h_y, *h_acceleration, *h_velocity;

  double masses = 5.0;

  struct system h_system;

  // Allocation
  h_x = (double *)malloc (sizeof (double) * N);
  h_y = (double *)malloc (sizeof (double) * N);

  // Cuda Allocation
  hipMalloc (&d_x, sizeof (double) * N);
  hipMalloc (&d_y, sizeof (double) * N);
  hipMalloc (&d_acceleration, sizeof (double) * N);
  hipMalloc (&d_velocity, sizeof (double) * N);

  // Initialisation value between 0.0 & 1.0
  init(h_system, N);
  
  // Copy host to device
  hipMemcpy (d_x, h_x, sizeof (double) * N, hipMemcpyHostToDevice);
  hipMemcpy (d_y, h_y, sizeof (double) * N, hipMemcpyHostToDevice);
  hipMemcpy (d_acceleration, h_acceleration, sizeof (double) * N, hipMemcpyHostToDevice);
  hipMemcpy (d_velocity, h_velocity, sizeof (double) * N, hipMemcpyHostToDevice);

  // device system initialisation
  struct system d_system = {d_x, d_y, d_acceleration, d_velocity, masses};

  // Kernel
  nbody_update<<<64, 64>>> (d_system, N);

  // Copy device to host
  hipMemcpy (h_x, d_x, sizeof (double) * N, hipMemcpyDeviceToHost);
  hipMemcpy (h_y, d_y, sizeof (double) * N, hipMemcpyDeviceToHost);
  hipMemcpy (h_acceleration, d_acceleration, sizeof (double) * N, hipMemcpyDeviceToHost);
  hipMemcpy (h_velocity, d_velocity, sizeof (double) * N, hipMemcpyDeviceToHost);

  // Free device & host
  hipFree (d_x);
  hipFree (d_y);
  hipFree (d_acceleration);
  hipFree (d_velocity);

  free (h_x);
  free (h_y);
  free (h_acceleration);
  free (h_velocity);

  return 0;
}
