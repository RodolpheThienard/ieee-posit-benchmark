#include "hip/hip_runtime.h"
extern "C" {
  #include "../../include/utils.h"
}
  #include "../../include/kernels.h"
  #include "../../include/hip/hip_runtime.h"

#define DRIVER_BODY(fn, ...)                                                  \
  {                                                                           \
    dim3 threadsPerBlock (32, 32);                                            \
    dim3 numBlocks (i / threadsPerBlock.x,                          \
                    i / threadsPerBlock.y);                         \
    hipEvent_t start_event, end_event;                                       \
    float elapsed;                                                            \
    for (int stability = 0; stability < 33; stability++)                      \
      {                                                                       \
        hipEventCreate (&start_event);                                       \
        hipEventCreate (&end_event);                                         \
        hipEventRecord (start_event, 0);                                     \
        for (int rep = 0; rep < bench.data->repetition; rep++)                      \
          {                                                                   \
            fn<<<numBlocks, threadsPerBlock>>> (__VA_ARGS__);                 \
          }                                                                   \
        hipEventRecord (end_event, 0);                                       \
        hipEventSynchronize (end_event);                                     \
        hipEventElapsedTime (&elapsed, start_event, end_event);              \
        elapsed *= 1e-3;                                                      \
        bench.data->samples[stability]                                       \
            = (double)elapsed / bench.data->repetition;                      \
      }                                                                       \
  }
  
__global__ void
inve_matrix_gauss_jordan2 (double *mat, double *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          double tmp = mat[i * n + i];
          ;
          // get pivot
          double pivot = tmp;

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              double coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}

__global__ void
dgemm2(double *A, double *B, double *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  double sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}
int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (double);
  struct bench bench = { data, accuracy, 100, 200, 100 };

  char buffer[1000];
  print_header_diff (buffer);
  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      bench.data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;

      double *a, *b, *c, *d, *d_a, *d_b, *d_c, *d_d;
      a = (double *)malloc (sizeof (double) * _matrix_size_2);
      b = (double *)malloc (sizeof (double) * _matrix_size_2);
      c = (double *)malloc (sizeof (double) * _matrix_size_2);
      d = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_d, _matrix_size_2 * sizeof (double));
      INIT (a, _matrix_size_2);
      INIT (b, _matrix_size_2);

      for (int i = 0; i < _matrix_size_2; i++)
      {
        a[i]=  drand48();
      }
      
      fprintf(stdout, "b device : %20.13lf\n", b[0]);
      fprintf(stdout, "a device : %20.13lf\n", a[0]);

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);


      inve_matrix_gauss_jordan2<<<32,32>>>(d_a, d_b, i);

      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      hipMemcpy (a, d_a, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);

      fprintf(stdout, "b device : %20.13lf\n", b[0]);
      fprintf(stdout, "a device : %20.13lf\n", a[0]);
      set_identity_matrix (c, i, i);
      hipMemcpy (d_c, c, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      dgemm2<<<32,32>>>(d_c, d_b, d_d, i);

      inve_matrix_gauss_jordan2<<<32,32>>>(d_d, d_b, i);

      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      hipMemcpy (a, d_a, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);

      fprintf(stdout, "b device : %20.13lf\n", b[0]);
      fprintf(stdout, "a device : %20.13lf\n", a[0]);
      formatting_data (bench.data);
      print_data_benchmark ("inversion gauss jordan", bench.data, buffer);

      bench.accuracy->accuracy = compute_err_accuracy (a, b, i);
      bench.accuracy->RMS = RMS (a, b, i);
      bench.accuracy->forward_error = forward_error (a, b, i);

      print_diff_accuracy ("Comparaison", buffer, &bench, bench.data);
      free (a);
      free (b);
      free (c);
      free (d);
      save_data (NULL, buffer);
    }

  free (data);
  return 0;
}
