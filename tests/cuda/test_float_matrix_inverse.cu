extern "C"
{
#include "../../include/utils.h"
}
#include "../../include/driver.h"
#include "../../include/hip/hip_runtime.h"
#include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (float);
  struct bench_s bench = { data, accuracy, 100, 200, 100 };

  char buffer[1000];
  print_header_diff (buffer);
  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      bench.data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;

      float *a, *b, *c, *d, *d_a, *d_b;
      double *aa,*bb;
      a = (float *)malloc (sizeof (float) * _matrix_size_2);
      b = (float *)malloc (sizeof (float) * _matrix_size_2);
      aa = (double *)malloc (sizeof (double) * _matrix_size_2);
      bb = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (float));
      INIT (aa, _matrix_size_2);

      for (int i = 0; i < _matrix_size_2; i++)
        {
          a[i] = aa[i];
        }


      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);

      driver_inverse_gauss_jordan (inve_matrix_gauss_jordan, i, d_a, d_b,
                                   &bench);

      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (float),
                  hipMemcpyDeviceToHost);

      inve_matrix_gauss_jordan_double(aa, bb, i);
                
      fprintf (stdout, "b device : %20.13lf;%20.13lf;\n", (double)b[0], bb[0]);
      fprintf (stdout, "a device : %20.13lf;%20.13lf;\n", (double)a[0], aa[0]);

      free (a);
      free (b);
      save_data (NULL, buffer);
    }

  free (data);
  return 0;
}
