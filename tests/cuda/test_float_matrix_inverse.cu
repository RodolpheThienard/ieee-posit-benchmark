extern "C"
{
  #include "../../include/utils.h"
  #include "../../include/host_kernels.h"

}
#include "../../include/driver.h"
#include "../../include/hip/hip_runtime.h"
#include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (float);
  struct bench_s bench = { data, accuracy, 100, 200, 100 };

  char buffer[1000];
  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      bench.data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;

      float *a, *b, *c, *d, *d_a, *d_b;
      double *aa, *bb;
      a = (float *)malloc (sizeof (float) * _matrix_size_2);
      b = (float *)malloc (sizeof (float) * _matrix_size_2);
      aa = (double *)malloc (sizeof (double) * _matrix_size_2);
      bb = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (float));
      INIT (aa, _matrix_size_2);

      for (int i = 0; i < _matrix_size_2; i++)
        {
          a[i] = aa[i];
        }
      set_identity_matrix (b, i, i);

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_b, b, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);

      for (int kk = 0; kk < i; kk++)
        {
          driver_inverse_gauss_jordan (inve_matrix_gauss_jordan_cuda, i, d_a, d_b,
                                       &bench);
        }

      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (float),
                  hipMemcpyDeviceToHost);
      hipMemcpy (a, d_a, _matrix_size_2 * sizeof (float),
                  hipMemcpyDeviceToHost);

      host_inve_matrix_gauss_jordan (aa, bb, i);


      free (a);
      free (b);
      save_data (NULL, buffer);
    }

  free (data);
  return 0;
}
