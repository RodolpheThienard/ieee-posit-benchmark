
extern "C"
{
#include "../../include/utils.h"
#include "../../include/host_kernels.h"
}
#include "../../include/kernels.h"
#include "../../include/hip/hip_runtime.h"
#include "../../include/driver.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char buffer[1000];
  data->type = sizeof (double);
  struct bench_s bench = { data, accuracy, 1024, 2048, 128 };

  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      int _matrix_size_2 = i * i;
      bench.data->matrice_size = _matrix_size_2;

      float *a, *b, *c, *c_host, *d_a, *d_b, *d_c;
      a = (float *)malloc (sizeof (float) * _matrix_size_2);
      b = (float *)malloc (sizeof (float) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (float));


      double *aa,*bb, *c_device;
      aa = (double *)malloc (sizeof (double) * _matrix_size_2);
      bb = (double *)malloc (sizeof (double) * _matrix_size_2);
      c_device = (double *)malloc (sizeof (double) * _matrix_size_2);
      INIT (aa, _matrix_size_2);
      for (int ll = 0; ll < _matrix_size_2; ll++)
      {
        a[ll] = aa[ll];
      }

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);

      driver_pi_approximation (pi_approximation, _matrix_size_2, d_a, d_b, &bench);

      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (float),
                  hipMemcpyDeviceToHost);

      host_pi_approximation (aa, bb, _matrix_size_2);


      conversion_into_double(b, c_device, _matrix_size_2);

      driver_accuracy (_matrix_size_2, bb, c_device, &bench);
      print_data_accuracy (buffer, bench.accuracy);

      free (a);
      free (b);
      hipFree (d_a);
      hipFree (d_b);
    }

  save_data (NULL, buffer);
  free (data);
  free (accuracy);
  return error_accuracy(&bench);
}



