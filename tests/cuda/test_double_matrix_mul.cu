#include "hip/hip_runtime.h"
#include <cstring>
extern "C"
{
#include "../../include/utils.h"
}
#include "../../include/kernels.h"
#include "../../include/hip/hip_runtime.h"

extern __global__ void dgemm (double *matrix1, double *matrix2, double *matrix3, int size);

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (double);
  struct bench bench = { data, accuracy, 100, 1000, 100 };

  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      bench.data->matrice_size = _matrix_size;
      int _matrix_size_2 = _matrix_size * _matrix_size;

      double *a, *b, *c, *c_host, *d_a, *d_b, *d_c;
      a = (double *)malloc (sizeof (double) * _matrix_size_2);
      b = (double *)malloc (sizeof (double) * _matrix_size_2);
      c = (double *)malloc (sizeof (double) * _matrix_size_2);
      c_host = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (double));

      INIT (a, _matrix_size_2);
      INIT (b, _matrix_size_2);

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_b, b, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);

      dgemm<<<32,32>>>(d_a, d_b, d_c, i);

      hipMemcpy (c, d_c, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);

      ieee_64bits_gemm(a, b, c_host, i);

      double err = RMS(c, c_host, i);

      fprintf(stdout, "double matrix mul err : %le\n", err);
      fprintf(stdout, "c device : %20.13lf\n", c[0]);
      fprintf(stdout, "c_host   : %20.13lf\n", c_host[0]);
      free (a);
      free (b);
      free (c);
      free (c_host);
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
    }

  free (data);
  return 0;
}
