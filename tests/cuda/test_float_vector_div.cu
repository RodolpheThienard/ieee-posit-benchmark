extern "C"
{
#include "../../include/utils.h"
#include "../../include/host_kernels.h"
}
#include "../../include/kernels.h"
#include "../../include/hip/hip_runtime.h"
#include "../../include/driver.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char buffer[1000];
  data->type = sizeof (double);
  struct bench_s bench = { data, accuracy, 1024, 2048, 128 };

  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      int _matrix_size_2 = i * i;
      bench.data->matrice_size = _matrix_size_2;

      float *a, *b, *c, *c_host, *d_a, *d_b, *d_c;
      a = (float *)malloc (sizeof (float) * _matrix_size_2);
      b = (float *)malloc (sizeof (float) * _matrix_size_2);
      c = (float *)malloc (sizeof (float) * _matrix_size_2);
      c_host = (float *)malloc (sizeof (float) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (float));

      // INIT (a, _matrix_size_2);
      // INIT (b, _matrix_size_2);

      double *aa,*bb,*cc, *c_device;
      aa = (double *)malloc (sizeof (double) * _matrix_size_2);
      bb = (double *)malloc (sizeof (double) * _matrix_size_2);
      cc = (double *)malloc (sizeof (double) * _matrix_size_2);
      c_device = (double *)malloc (sizeof (double) * _matrix_size_2);
      INIT (aa, _matrix_size_2);
      for (int ll = 0; ll < _matrix_size_2; ll++)
      {
        a[ll] = aa[ll];
        b[ll] = bb[ll];
      }

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_b, b, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);

      driver_vector_div (vector_div, _matrix_size_2, d_a, d_b, d_c, &bench);

      hipMemcpy (c, d_c, _matrix_size_2 * sizeof (float),
                  hipMemcpyDeviceToHost);

      host_vector_div (aa, bb, cc, _matrix_size);


      conversion_into_double(c, c_device, _matrix_size_2);

      driver_accuracy (_matrix_size_2, cc, c_device, &bench);
      print_data_accuracy (buffer, bench.accuracy);

      free (a);
      free (b);
      free (c);
      free (c_host);
      hipFree (d_a);
      hipFree (d_b);
      hipFree (d_c);
    }

  save_data (NULL, buffer);
  free (data);
  free (accuracy);
  return 0;
}


