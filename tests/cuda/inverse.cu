#include "hip/hip_runtime.h"
extern "C" {
  #include "../../include/utils.h"
}
  #include "../../include/kernels.h"
  #include "../../include/hip/hip_runtime.h"

#define DRIVER_BODY(fn, ...)                                                  \
  {                                                                           \
    dim3 threadsPerBlock (32, 32);                                            \
    dim3 numBlocks (i / threadsPerBlock.x,                          \
                    i / threadsPerBlock.y);                         \
    hipEvent_t start_event, end_event;                                       \
    float elapsed;                                                            \
    for (int stability = 0; stability < 33; stability++)                      \
      {                                                                       \
        hipEventCreate (&start_event);                                       \
        hipEventCreate (&end_event);                                         \
        hipEventRecord (start_event, 0);                                     \
        for (int rep = 0; rep < bench.data->repetition; rep++)                      \
          {                                                                   \
            fn<<<numBlocks, threadsPerBlock>>> (__VA_ARGS__);                 \
          }                                                                   \
        hipEventRecord (end_event, 0);                                       \
        hipEventSynchronize (end_event);                                     \
        hipEventElapsedTime (&elapsed, start_event, end_event);              \
        elapsed *= 1e-3;                                                      \
        bench.data->samples[stability]                                       \
            = (double)elapsed / bench.data->repetition;                      \
      }                                                                       \
  }
  
__global__ void
inve_matrix_gauss_jordan2 (double *mat, double *inv, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n)
    {
      for (int i = 0; i < n; i++)
        {
          double tmp = mat[i * n + i];
          ;
          // get pivot
          double pivot = tmp;

          // Dividing by pivot
          mat[i * n + tid] /= pivot;
          inv[i * n + tid] /= pivot;

          __syncthreads ();

          // remove other elements
          if (tid != i)
            {
              double coeff = mat[tid * n + i];
              for (int j = 0; j < n; j++)
                {
                  mat[tid * n + j] -= coeff * mat[i * n + j];
                  inv[tid * n + j] -= coeff * inv[i * n + j];
                }
            }

          __syncthreads ();
        }
    }
}

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (double);
  struct bench bench = { data, accuracy, 100, 3000, 100 };

  char buffer[1000];
  print_header_diff (buffer);
  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      bench.data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;

      double *a, *b, *c, *d, *d_a, *d_b, *d_c, *d_d;
      a = (double *)malloc (sizeof (double) * _matrix_size_2);
      b = (double *)malloc (sizeof (double) * _matrix_size_2);
      c = (double *)malloc (sizeof (double) * _matrix_size_2);
      d = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_d, _matrix_size_2 * sizeof (double));
      INIT (a, _matrix_size_2);
      INIT (b, _matrix_size_2);

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);

      DRIVER_BODY (inve_matrix_gauss_jordan2, d_a, d_b, i);

      set_identity_matrix (c, i, i);
      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_c, c, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      ieee_64bits_gemm_jik (d_c, d_b, d_d, i);

      DRIVER_BODY (inve_matrix_gauss_jordan2, d_d, d_b, i);

      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      hipMemcpy (a, d_a, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      formatting_data (bench.data);
      print_data_benchmark ("inversion gauss jordan", bench.data, buffer);

      bench.accuracy->accuracy = compute_err_accuracy (a, b, i);
      bench.accuracy->RMS = RMS (a, b, i);
      bench.accuracy->forward_error = forward_error (a, b, i);

      print_diff_accuracy ("Comparaison", buffer, &bench, bench.data);
      free (a);
      free (b);
      free (c);
      free (d);
      save_data (NULL, buffer);
    }

  free (data);
  return 0;
}
