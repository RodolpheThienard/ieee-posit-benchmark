extern "C" {
  #include "../../include/utils.h"
}
  #include "../../include/driver.h"
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  long _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (double);
  struct bench bench = { data, accuracy, 100, 200, 100 };

               
  benchmark ("sinus_libmath", "sinus_macclaurin", NULL, sinus_libmath,
             sinus_maclaurin, &bench, KERNEL2, _matrix_size);
  // benchmark ("SQRT libmath", "SQRT newton", NULL, sqrt_libmath,
  //            square_root_newton_raphson, &bench, KERNEL2, _matrix_size);
  // benchmark ("Log libmath", "Log Taylor", NULL, log_libmath,
  // logarithm_taylor,
  //            &bench, KERNEL2, _matrix_size);

  // benchmark ("FP32", "FP64", NULL, conversion_double_float,
  // useless_function,
  //            &bench, KERNEL2, _matrix_size);
  // benchmark ("Monte-carlo-FP32", "Monte-carlo-FP64", NULL,
  //            monte_carlo_option_pricing_fp32,
  //            monte_carlo_option_pricing_fp64, &bench, KERNEL2,
  //            _matrix_size);
  // benchmark ("PI FP64", "PI FP32", NULL, pi_approximation_fp64,
  //            pi_approximation_fp32, &bench, KERNEL2, _matrix_size);
  // benchmark ("PI FP64", "Real PI", NULL, pi_approximation_fp64, real_pi,
  //            &bench, KERNEL2, _matrix_size);

  // INVERSE MATRIX
  // benchmark ("gauss jordan", NULL, "gauss.dat", inve_matrix_gauss_jordan,
  // NULL, &bench, INVERSION, _matrix_size);

  // KERNEL 1
  // benchmark ("SQRT BLAS", NULL, NULL, ieee_64bits_sqrt, NULL, &bench,
  // KERNEL1,
  //            _matrix_size);

  free (data);
  return 0;
}
