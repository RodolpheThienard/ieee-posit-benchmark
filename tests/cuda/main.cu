#include "hip/hip_runtime.h"
extern "C" {
  #include "../../include/utils.h"
}
  #include "../../include/driver.h"
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data*)malloc(sizeof(struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy*)malloc(sizeof(struct accuracy));
  
  uint64_t _matrix_size = atoll (argv[1]);
  uint64_t _repetition = atoll (argv[2]);

  uint64_t _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char *output_buffer;
  output_buffer = (char *)malloc (sizeof (char) * 200 * 11);
  print_header_benchmark (output_buffer, _matrix_size_2 * 2);
  data->type = sizeof (double);

  // driver_cuda_fp64_benchmark ("cuda gemm ijk", output_buffer, cuda_64bits_gemm_ijk, data,
  //                             _matrix_size);
  // driver_cuda_fp64_benchmark ("cuda gemm jik", output_buffer, cuda_64bits_gemm_jik, data, 
  //                _matrix_size);
  // driver_cuda_fp64_benchmark ("cuda gemm bloc", output_buffer, cuda_64bits_gemm_bloc, data, 
  //                _matrix_size);

  print_header_accuracy(output_buffer);

  // driver_cuda_inv_matrix_accuracy("Inversion Gauss Jordan", output_buffer , cuda_inve_matrix_gauss_jordan, accuracy, _matrix_size);
  driver_compare_accuracy("Log vs log Taylor", output_buffer , logarithm_taylor, log_libmath, accuracy, _matrix_size);

  save_data (NULL, output_buffer);

 }
