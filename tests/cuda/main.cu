extern "C" {
  #include "../../include/utils.h"
  #include "../../include/driver.h"
}
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data*)malloc(sizeof(struct data));
  
  uint64_t _matrix_size = atoll (argv[1]);
  uint64_t _repetition = atoll (argv[2]);

  uint64_t _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char *output_buffer;
  output_buffer = (char *)malloc (sizeof (char) * 200 * 11);
  print_header_benchmark (output_buffer, _matrix_size_2 * 2);
  data->type = sizeof (double);

  driver_cuda_fp64_benchmark ("cuda gemm ijk", output_buffer, cuda_64bits_gemm_ijk, data,
                              _matrix_size);
  driver_cuda_fp64_benchmark ("cuda gemm jik", output_buffer, cuda_64bits_gemm_jik, data, 
                 _matrix_size);
  driver_cuda_fp64_benchmark ("cuda gemm bloc", output_buffer, cuda_64bits_gemm_bloc, data, 
                 _matrix_size);

  save_data (NULL, output_buffer);

 }
