extern "C" {
  #include "../../include/utils.h"
  #include "../../include/driver.h"
}
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  int N = 1 << 20;
  double *A, *B, *C, *d_A, *d_B, *d_C;

  N = (argc < 2) ? 1000 : atoi (argv[1]);

  // Memory allocation
  A = (double *)malloc (sizeof (double) * N * N);
  B = (double *)malloc (sizeof (double) * N * N);
  C = (double *)malloc (sizeof (double) * N * N);


  INIT (A, N);
  INIT (B, N);
  INIT (C, N);

  hipblasHandle_t handle;
  hipblasCreate (&handle);
  hipMalloc (&d_A, N * N * sizeof (double));
  hipMalloc (&d_B, N * N * sizeof (double));
  hipMalloc (&d_C, N * N * sizeof (double));

  hipMemcpy (d_A, A, N * N * sizeof (double), hipMemcpyHostToDevice);
  hipMemcpy (d_B, B, N * N * sizeof (double), hipMemcpyHostToDevice);


  struct data *data;
  data = (struct data*)malloc(sizeof(struct data));
  
  uint64_t _matrix_size = atoll (argv[1]);
  uint64_t _repetition = atoll (argv[2]);

  uint64_t _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  print_header(_matrix_size_2);

  data->type = 8;
  driver_cuda_fp64 ("cuda gemm ijk", cuda_64bits_gemm_ijk, data, d_A, d_B, d_C,
                    N);
  driver_cuda_fp64 ("cuda gemm jik", cuda_64bits_gemm_jik, data, d_A, d_B, d_C,
                    N);
  driver_cuda_fp64 ("cuda gemm bloc", cuda_64bits_gemm_bloc, data, d_A, d_B, d_C,
                    N);

  hipFree (d_A);
  hipFree (d_B);
  hipFree (d_C);
  free (A);
  free (B);
  free (C);
}
