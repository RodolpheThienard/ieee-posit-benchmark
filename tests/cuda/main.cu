extern "C" {
  #include "../../include/utils.h"
  #include "../../include/driver.h"
}
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  int N = 1 << 20;
  double *A, *B, *C, *d_A, *d_B, *d_C;

  N = (argc < 2) ? 1000 : atoi (argv[1]);

  // Memory allocation
  A = (double *)malloc (sizeof (double) * N * N);
  B = (double *)malloc (sizeof (double) * N * N);
  C = (double *)malloc (sizeof (double) * N * N);


  INIT (A, N);
  INIT (B, N);
  INIT (C, N);

  hipblasHandle_t handle;
  hipblasCreate (&handle);
  hipMalloc (&d_A, N * N * sizeof (double));
  hipMalloc (&d_B, N * N * sizeof (double));
  hipMalloc (&d_C, N * N * sizeof (double));

  hipMemcpy (d_A, A, N * N * sizeof (double), hipMemcpyHostToDevice);
  hipMemcpy (d_B, B, N * N * sizeof (double), hipMemcpyHostToDevice);


  struct data *data;
  data = (struct data*)malloc(sizeof(struct data));
  
  uint64_t _matrix_size = atoll (argv[1]);
  uint64_t _repetition = atoll (argv[2]);

  uint64_t _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char *output_buffer;
  output_buffer = (char*)malloc (sizeof (char) * 200 * 11);
  print_header (output_buffer, _matrix_size_2 * 2);

  data->type = sizeof(double);
  driver_cuda_fp64 ("cuda gemm ijk", output_buffer, cuda_64bits_gemm_ijk, data, d_A, d_B, d_C,
                    N);
  driver_cuda_fp64 ("cuda gemm jik", output_buffer, cuda_64bits_gemm_jik, data, d_A, d_B, d_C,
                    N);
  driver_cuda_fp64 ("cuda gemm bloc", output_buffer, cuda_64bits_gemm_bloc, data, d_A, d_B, d_C,
                    N);

  save_data ("test.csv", output_buffer);
                  

  hipFree (d_A);
  hipFree (d_B);
  hipFree (d_C);
  free (A);
  free (B);
  free (C);
}
