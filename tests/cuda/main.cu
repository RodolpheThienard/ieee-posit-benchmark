#include "hip/hip_runtime.h"
extern "C" {
  #include "../../include/utils.h"
}
  #include "../../include/driver.h"
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data*)malloc(sizeof(struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy*)malloc(sizeof(struct accuracy));
  
  long _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char *output_buffer;
  output_buffer = (char *)malloc (sizeof (char) * 200 * 11);
  print_header_benchmark (output_buffer, _matrix_size_2 * 2);
  data->type = sizeof (double);

  driver_fp64_benchmark ("ieee gemm ijk", output_buffer, ieee_64bits_gemm_ijk, data,
                              _matrix_size);
  driver_fp64_benchmark ("ieee gemm jik", output_buffer, ieee_64bits_gemm_jik, data, 
                 _matrix_size);
  driver_fp64_benchmark ("ieee gemm bloc", output_buffer, ieee_64bits_gemm_bloc, data, 
                 _matrix_size);

  print_header_accuracy(output_buffer);

  driver_inv_matrix_accuracy("Inversion Gauss Jordan", output_buffer , inve_matrix_gauss_jordan, accuracy, _matrix_size);
  driver_compare_accuracy("Log vs log Taylor", output_buffer , logarithm_taylor, log_libmath, accuracy, _matrix_size);

  save_data (NULL, output_buffer);

 }
