extern "C" {
  #include "../../include/utils.h"
  // #include "../../include/driver.h"
}
  #include "../../include/hip/hip_runtime.h"
  #include "../../include/kernels.h"

#define DRIVER_BODY_BENCHMARK_CUDA(kernel, ...)                                         \
  dim3 threadsPerBlock (32, 32);                                              \
  dim3 numBlocks (matrix_size / threadsPerBlock.x,                            \
                  matrix_size / threadsPerBlock.y);                           \
  hipEvent_t start_event, end_event;                                         \
  float elapsed;                                                              \
  for (uint32_t stability = 0; stability < 33; stability++)                   \
    {                                                                         \
      hipEventCreate (&start_event);                                         \
      hipEventCreate (&end_event);                                           \
      hipEventRecord (start_event, 0);                                       \
      for (uint32_t rep = 0; rep < data->repetition; rep++)                   \
        {                                                                     \
          kernel<<<numBlocks, threadsPerBlock>>> (__VA_ARGS__);               \
        }                                                                     \
      hipEventRecord (end_event, 0);                                         \
      hipEventSynchronize (end_event);                                       \
      hipEventElapsedTime (&elapsed, start_event, end_event);                \
      elapsed *= 1e-3;                                                        \
      data->samples[stability] = (double)elapsed / data->repetition;          \
    }                                                                         

void
driver_cuda_fp64_benchmark (char *title, char *buffer,
                            void (*kernel) (double *, double *, double *, int),
                            struct data *data, int matrix_size)
{
  double *A, *B, *C, *d_A, *d_B, *d_C;
  long _matrix_size_2 = matrix_size * matrix_size;
  A = (double *)malloc (sizeof (double) * _matrix_size_2);
  B = (double *)malloc (sizeof (double) * _matrix_size_2);
  C = (double *)malloc (sizeof (double) * _matrix_size_2);

  INIT (A, _matrix_size_2);
  INIT (B, _matrix_size_2);
  INIT (C, _matrix_size_2);

  hipblasHandle_t handle;
  hipblasCreate (&handle);
  hipMalloc (&d_A, _matrix_size_2 * sizeof (double));
  hipMalloc (&d_B, _matrix_size_2 * sizeof (double));
  hipMalloc (&d_C, _matrix_size_2 * sizeof (double));

  hipMemcpy (d_A, A, _matrix_size_2 * sizeof (double),
              hipMemcpyHostToDevice);
  hipMemcpy (d_B, B, _matrix_size_2 * sizeof (double),
              hipMemcpyHostToDevice);

  DRIVER_BODY_BENCHMARK_CUDA (kernel, d_A, d_B, d_C, matrix_size);
  formatting_data (data);
  print_data_benchmark (title, data, buffer);

  hipFree (d_A);
  hipFree (d_B);
  hipFree (d_C);
  free (A);
  free (B);
  free (C);
}
int
main (int argc, char *argv[])
{

  struct data *data;
  data = (struct data*)malloc(sizeof(struct data));
  
  uint64_t _matrix_size = atoll (argv[1]);
  uint64_t _repetition = atoll (argv[2]);

  uint64_t _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  char *output_buffer;
  output_buffer = (char *)malloc (sizeof (char) * 200 * 11);
  print_header_benchmark (output_buffer, _matrix_size_2 * 2);
  data->type = sizeof (double);
  char test[] = "cuda gemm ijk";

  driver_cuda_fp64_benchmark (test, output_buffer, cuda_64bits_gemm_ijk, data,
                              _matrix_size);
  // driver_cuda_fp64_benchmark ("cuda gemm jik", output_buffer, cuda_64bits_gemm_jik, data, 
  //                // N);
  // driver_cuda_fp64_benchmark ("cuda gemm bloc", output_buffer, cuda_64bits_gemm_bloc, data, 
  //                // N);

  save_data (NULL, output_buffer);

 }
