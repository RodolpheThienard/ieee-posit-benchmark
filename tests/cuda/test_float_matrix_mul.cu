#include "hip/hip_runtime.h"
extern "C"
{
#include "../../include/utils.h"
}
#include "../../include/kernels.h"
#include "../../include/hip/hip_runtime.h"
#include "../../include/driver.h"

float
compute_err_accuracy_float (float *a, float *b, int n)
{
  int i = 0;
  float err = 0.0;
  for (i = 0; i < n; i++)
    {
      err += (a[i] - b[i])*(a[i] - b[i]);
    }
   err /= n;
  return sqrt (err);
}

float
RMS_float (float *a, float *b, int n)
{
  int i = 0;
  float err = 0.0;
  for (i = 0; i < n; i++)
    {
      err += (a[i] - b[i]) * (a[i] - b[i]);
    }
  err /= n;
  return sqrt (err);
}
int
main (int argc, char *argv[])
{
  struct data *data;
  data = (struct data *)malloc (sizeof (struct data));
  struct accuracy *accuracy;
  accuracy = (struct accuracy *)malloc (sizeof (struct accuracy));

  int _matrix_size = atoll (argv[1]);
  long _repetition = atoll (argv[2]);

  long _matrix_size_2 = _matrix_size * _matrix_size;
  data->matrice_size = _matrix_size_2;
  data->repetition = _repetition;

  // ≃ 200 per kernel
  data->type = sizeof (double);
  struct bench_s bench = { data, accuracy, 100, 1000, 100 };

  for (int i = bench.start_size; i < bench.end_size; i += bench.step_size)
    {
      _matrix_size = i;
      bench.data->matrice_size = _matrix_size;
      int _matrix_size_2 = _matrix_size * _matrix_size;

      float *a, *b, *c, *c_host, *d_a, *d_b, *d_c;
      a = (float *)malloc (sizeof (float) * _matrix_size_2);
      b = (float *)malloc (sizeof (float) * _matrix_size_2);
      c = (float *)malloc (sizeof (float) * _matrix_size_2);
      c_host = (float *)malloc (sizeof (float) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (float));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (float));

      INIT (a, _matrix_size_2);
      INIT (b, _matrix_size_2);

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_b, b, _matrix_size_2 * sizeof (float),
                  hipMemcpyHostToDevice);

      driver_sgemm(sgemm, _matrix_size, d_a,d_b,d_c,&bench);

      hipMemcpy (c, d_c, _matrix_size_2 * sizeof (float),
                  hipMemcpyDeviceToHost);

      ieee_32bits_gemm(a, b, c_host, i);

      // driver_accuracy_32bits(_matrix_size_2, c_host, d_c, &bench);

      float err = compute_err_accuracy_float(c, c_host, i);
      float rms = RMS_float(c, c_host, i);

      fprintf(stdout, "float matrix mul err : %le; rms : %le\n", err, rms);
      free (a);
      free (b);
      free (c);
      free (c_host);
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
    }

  free (data);
  return 0;
}
