#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstddef>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#define max(a, b) ((a) < (b) ? (b) : (a))
#define gettime(t) clock_gettime (CLOCK_MONOTONIC_RAW, t)
#define get_sub_seconde(t) (1e-9 * (float)t.tv_nsec)


void
init (float *A, float *B, float *C, int size)
{
  int i = 0, j = 0;

  srand (2019);

  for (i = 0; i < size; i++)
    {
      for (j = 0; j < size; j++)
        {
          A[i * size + j] = (float)(rand () % 10) + 0.01 * (rand () % 5);
          B[i * size + j] = (float)(rand () % 10) + 0.01 * (rand () % 5);
          C[i * size + j] = 0.0;
        }
    }
}
__global__ void
sgemm_jik (float *A, float *B, float *C, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}
__global__ void
sgemm_ijk (float *A, float *B, float *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Col
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.;
  for (int k = 0; k < N; k++)
    {
      sum += A[i * N + k] * B[k * N + j];
    }
  C[i * N + j] = sum;
}

__global__ void 
sgemm_bloc(float *A, float *B, float *C, int N)
{
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     int j = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ float a[32][32];
    __shared__ float b[32][32];

    float sum = 0;
    for (int k = 0; k < N; k += 32) {
        a[threadIdx.y][threadIdx.x] = A[j * N + k + threadIdx.x];
        b[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + i];

        __syncthreads();

        for (int l = 0; l < 32; l++)
            sum += a[threadIdx.y][l] * b[l][threadIdx.x];

        __syncthreads();
    }
    C[j * N + i] = sum;
}
float
get_elapsedtime (void)
{
  struct timespec st;
  int err = gettime (&st);
  if (err != 0)
    return 0;
  return (float)st.tv_sec + get_sub_seconde (st);
}

int
main (int argc, char *argv[])
{
  int N = 1 << 20;
  float *A, *B, *C, *d_A, *d_B, *d_C, t1, t0;

  N = (argc < 2) ? 1000 : atoi (argv[1]);
  // fprintf (stdout, "Matrix Multiplication\n  Size: %dx%d\n", N, N);

  // Memory allocation
  A = (float *)malloc (sizeof (float) * N * N);
  B = (float *)malloc (sizeof (float) * N * N);
  C = (float *)malloc (sizeof (float) * N * N);

  // float *ms = (float*)malloc(sizeof(float));
  float ms=0;
  
  init (A, B, C, N);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipMalloc (&d_A, N * N * sizeof (float));
  hipMalloc (&d_B, N * N * sizeof (float));
  hipMalloc (&d_C, N * N * sizeof (float));

  hipMemcpy (d_A, A, N * N * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpy (d_B, B, N * N * sizeof (float), hipMemcpyHostToDevice);
  float un = 1;

  // Kernel invocation
  dim3 threadsPerBlock (32, 32);
  dim3 numBlocks (N / threadsPerBlock.x, N / threadsPerBlock.y);

  // Perform SAXPY on 1M elements
  hipEvent_t start_event, end_event;
  hipEventCreate(&start_event);
  hipEventCreate(&end_event);

  hipEventRecord(start_event, 0);
  sgemm_bloc<<<numBlocks, threadsPerBlock>>> (d_A, d_B, d_C, N);
  hipEventRecord(end_event, 0);
  hipEventSynchronize(end_event);

  hipError_t err = hipGetLastError();
  hipMemcpy (C, d_C, N * sizeof (float), hipMemcpyDeviceToHost);

  hipEventElapsedTime(&ms, start_event, end_event);
  // Pretty print
  long long N_u64 = (long long)N;
  long long nb_op = N_u64 * N_u64 * N_u64;
  // fprintf (stdout, "Performance results: \n");
  // fprintf (stdout, "  Time: %e ms\n", ms);
  // fprintf (stdout, "  MFlops: %.2f\n", (nb_op / ms*1e3) * 1E-6);
  fprintf(stdout, "%d; %f; %.2f\n",N, ms, ((nb_op/(ms*1e-3))*1e-9));

  hipFree (d_A);
  hipFree (d_B);
  hipFree (d_C);
  free (A);
  free (B);
  free (C);
}
