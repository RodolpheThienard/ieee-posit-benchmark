extern "C" {
  #include "../include/driver.h"
}
  #include "../include/hip/hip_runtime.h"

#define DRIVER_BODY_CUDA(kernel, ...)                                         \
  dim3 threadsPerBlock (32, 32);                                              \
  dim3 numBlocks (matrix_size / threadsPerBlock.x,                            \
                  matrix_size / threadsPerBlock.y);                           \
  hipEvent_t start_event, end_event;                                         \
  float elapsed;                                                              \
  for (uint32_t stability = 0; stability < 33; stability++)                   \
    {                                                                         \
      hipEventCreate (&start_event);                                         \
      hipEventCreate (&end_event);                                           \
      hipEventRecord (start_event, 0);                                       \
      for (uint32_t rep = 0; rep < data->repetition; rep++)                   \
        {                                                                     \
          kernel<<<numBlocks, threadsPerBlock>>> (__VA_ARGS__);               \
        }                                                                     \
      hipEventRecord (end_event, 0);                                         \
      hipEventSynchronize (end_event);                                       \
      hipEventElapsedTime (&elapsed, start_event, end_event);                \
      elapsed *= 1e-3;                                                        \
      data->samples[stability] = (double)elapsed / data->repetition;          \
    }                                                                         \

  
void
driver_cuda_fp64 (char *title, void (*kernel) (double*, double*, double*, int), struct data *data,
                  double * a, double *b, double *c,
                  int matrix_size)
{
  DRIVER_BODY_CUDA (kernel, a, b, c, matrix_size);
  formatting_data(data);
  print_data(title, data);
}
