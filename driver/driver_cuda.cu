#include <hipblas.h>
#include <hip/hip_runtime.h>

#define DRIVER_BODY_CUDA(kernel, ...)                                         \
  hipEvent_t start_event[33], end_event[33];                                 \
  for (uint32_t stability = 0; stability < 33; stability++)                   \
    {                                                                         \
      hipEventCreate (&start_event[stability]);                              \
      hipEventCreate (&end_event[stability]);                                \
      hipEventRecord (start_event[stability], 0);                            \
      for (uint32_t rep = 0; rep < data->repetition; rep++)                   \
        {                                                                     \
          kernel (__VA_ARGS__);                                               \
        }                                                                     \
      hipEventRecord (end_event[stability], 0);                              \
      hipEventSynchronize (end_event[stability]);                            \
      data->samples[stability] = end_event - start_event;                     \
    }                                                                         \
  sort_double (data->samples);                                                \
  data->mean = mean (data->samples);                                          \
  data->stddev = stddev (data->samples, data->mean);                          \
  print_data (title, data);

