#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../include/driver.h"

#define DRIVER_BODY_CUDA(kernel, ...)                                         \
  hipEvent_t start_event[33], end_event[33];                                 \
  for (uint32_t stability = 0; stability < 33; stability++)                   \
    {                                                                         \
      hipEventCreate (&start_event[stability]);                              \
      hipEventCreate (&end_event[stability]);                                \
      hipEventRecord (start_event[stability], 0);                            \
      for (uint32_t rep = 0; rep < data->repetition; rep++)                   \
        {                                                                     \
          kernel<<<64,64>>> (__VA_ARGS__);                                    \
        }                                                                     \
      hipEventRecord (end_event[stability], 0);                              \
      hipEventSynchronize (end_event[stability]);                            \
      data->samples[stability] = end_event - start_event;                     \
    }                                                                         \
  sort_double (data->samples);                                                \
  data->mean = mean (data->samples);                                          \
  data->stddev = stddev (data->samples, data->mean);                          \
  print_data (title, data);

void
driver_cuda_fp64 (char *title, void (*kernel) (), struct data *data,
                  double *restrict a, double *restrict b, double *restrict c,
                  uint64_t matrix_size)
{
  DRIVER_BODY_CUDA (kernel, a, b, c, matrix_size);
}
