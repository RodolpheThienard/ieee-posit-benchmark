#include <cstdio>
extern "C" {
#include "../../include/utils.h"
}
#include "../../include/hip/hip_runtime.h"
#include "../../include/kernels.h"

#define DRIVER_BODY_ACCURACY_CUDA(kernel, ...)                                 \
  {                                                                            \
    dim3 threadsPerBlock(32, 32);                                              \
    dim3 numBlocks(matrix_size / threadsPerBlock.x,                            \
                   matrix_size / threadsPerBlock.y);                           \
    kernel<<<numBlocks, threadsPerBlock>>>(__VA_ARGS__);                       \
  }

#define DRIVER_BODY_COMPARE_ACCURACY_CUDA(fn, fn2, a, b, size)                 \
  {                                                                            \
    dim3 threadsPerBlock(32, 32);                                              \
    dim3 numBlocks(matrix_size / threadsPerBlock.x,                            \
                   matrix_size / threadsPerBlock.y);                           \
    fn<<<numBlocks, threadsPerBlock>>>(a, size);                               \
    fn2<<<numBlocks, threadsPerBlock>>>(b, size);                              \
  }

/* driver for inv matrix computation
   Alloc and init matrix
   compute with given kernel
   compute identity matrix from A * A⁻¹
   check error between computed identity matrix and real identity matrix */
void driver_cuda_inv_matrix_accuracy(char *title, char *buffer,
                                     void (*kernel)(double *, double *, int),
                                     struct accuracy *accuracy,
                                     int matrix_size) {
  // initialisation matrix
  double *A, *B, *C, *D, *d_A, *d_B, *d_C, *d_D;
  int _matrix_size_2 = matrix_size * matrix_size;
  A = (double *)malloc(sizeof(double) * _matrix_size_2);
  B = (double *)malloc(sizeof(double) * _matrix_size_2);
  C = (double *)malloc(sizeof(double) * _matrix_size_2);
  D = (double *)malloc(sizeof(double) * _matrix_size_2);

  INIT(A, _matrix_size_2);
  INIT(B, _matrix_size_2);

  hipMalloc(&d_A, _matrix_size_2 * sizeof(double));
  hipMalloc(&d_B, _matrix_size_2 * sizeof(double));
  hipMalloc(&d_C, _matrix_size_2 * sizeof(double));
  hipMalloc(&d_D, _matrix_size_2 * sizeof(double));

  hipMemcpy(d_A, A, _matrix_size_2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, _matrix_size_2 * sizeof(double), hipMemcpyHostToDevice);
  fprintf(stdout, "ICI\n");
  DRIVER_BODY_ACCURACY_CUDA(kernel, d_A, d_B, matrix_size);

  set_identity_matrix(C, matrix_size, matrix_size);

  hipMemcpy(d_C, C, _matrix_size_2 * sizeof(double), hipMemcpyHostToDevice);
  ieee_64bits_gemm_bloc(C, B, D, matrix_size);

  DRIVER_BODY_ACCURACY_CUDA(kernel, d_D, d_B, matrix_size);
  hipMemcpy(A, d_A, _matrix_size_2 * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(B, d_B, _matrix_size_2 * sizeof(double), hipMemcpyDeviceToHost);

  accuracy->accuracy = compute_err_accuracy(A, B, _matrix_size_2);
  accuracy->RMS = RMS(A, B, _matrix_size_2);
  accuracy->forward_error = forward_error(A, B, _matrix_size_2);
  print_data_accuracy(title, buffer, accuracy);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);
  free(A);
  free(B);
  free(C);
  free(D);
}

/* driver to compare 2 vectors/matrix
   Alloc and init vector/matrix
   compute with given kernels
   check error between the two kernels  */
void driver_compare_accuracy(char *title, char *buffer, void (*kernel)(double *, int),
                                  void (*kernel_2)(double *, int), struct accuracy *accuracy,
                                  int matrix_size) {
  double *A, *B, *d_A, *d_B;
  int _matrix_size_2 = matrix_size * matrix_size;
  A = (double *)malloc(sizeof(double) * _matrix_size_2);
  B = (double *)malloc(sizeof(double) * _matrix_size_2);
  // INIT(A, matrix_size);

  hipMalloc(&d_A, _matrix_size_2 * sizeof(double));
  hipMalloc(&d_B, _matrix_size_2 * sizeof(double));
  // copying init values
  for (int i = 0; i < _matrix_size_2; i++) {
    A[i] += drand48();
    B[i] = A[i];
  }

  hipMemcpy(d_A, A, _matrix_size_2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, _matrix_size_2 * sizeof(double), hipMemcpyHostToDevice);

  DRIVER_BODY_COMPARE_ACCURACY_CUDA(kernel, kernel_2, d_A, d_B, _matrix_size_2);

  hipMemcpy(A, d_A, _matrix_size_2 * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(B, d_B, _matrix_size_2 * sizeof(double), hipMemcpyDeviceToHost);


  accuracy->accuracy = compute_err_accuracy(A, B, _matrix_size_2);
  accuracy->RMS = RMS(A, B, _matrix_size_2);
  accuracy->forward_error = forward_error(A, B, _matrix_size_2);

  print_data_accuracy(title, buffer, accuracy);

  hipFree(d_A);
  hipFree(d_B);
  free(A);
  free(B);
}
