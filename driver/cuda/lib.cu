extern "C" {
  #include "../../include/utils.h"
}
  #include "../../include/kernels.h"
  #include "../../include/hip/hip_runtime.h"

#define DRIVER_BODY(fn, ...)                                                  \
  {                                                                           \
    dim3 threadsPerBlock (32, 32);                                            \
    dim3 numBlocks (matrix_size / threadsPerBlock.x,                          \
                    matrix_size / threadsPerBlock.y);                         \
    hipEvent_t start_event, end_event;                                       \
    float elapsed;                                                            \
    for (int stability = 0; stability < 33; stability++)                      \
      {                                                                       \
        hipEventCreate (&start_event);                                       \
        hipEventCreate (&end_event);                                         \
        hipEventRecord (start_event, 0);                                     \
        for (int rep = 0; rep < bench->data->repetition; rep++)                      \
          {                                                                   \
            fn<<<numBlocks, threadsPerBlock>>> (__VA_ARGS__);                 \
          }                                                                   \
        hipEventRecord (end_event, 0);                                       \
        hipEventSynchronize (end_event);                                     \
        hipEventElapsedTime (&elapsed, start_event, end_event);              \
        elapsed *= 1e-3;                                                      \
        bench->data->samples[stability]                                       \
            = (double)elapsed / bench->data->repetition;                      \
      }                                                                       \
  }
  

void
kernel2 (char *name_kernel, char *name_kernel_2,
         void (*kernel) (double *, double *, int),
         void (*kernel_2) (double *, double *, int), char *filename,
         struct bench *bench, int matrix_size)
{
  char buffer[1000];
  print_header_diff (buffer);
  for (int i = bench->start_size; i < bench->end_size; i += bench->step_size)
    {
      int _matrix_size = i;
      bench->data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;
      double *a, *b, *c, *d, *d_a, *d_b, *d_c, *d_d;
      a = (double *)malloc (sizeof (double) * _matrix_size_2);
      b = (double *)malloc (sizeof (double) * _matrix_size_2);
      c = (double *)malloc (sizeof (double) * _matrix_size_2);
      d = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_d, _matrix_size_2 * sizeof (double));

      // copying init values
      for (int j = 0; j < _matrix_size; j++)
        {
          a[j] += drand48 ();
          c[j] = a[j];
        }

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_c, c, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      DRIVER_BODY (kernel, d_a, d_b, _matrix_size);
      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      formatting_data (bench->data);

      print_data_benchmark (name_kernel, bench->data, buffer);
      DRIVER_BODY (kernel_2, d_c, d_d, _matrix_size);
      hipMemcpy (d, d_c, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);

      struct data data_2 = *bench->data;
      formatting_data (bench->data);
      print_data_benchmark (name_kernel_2, bench->data, buffer);

      bench->accuracy->accuracy = compute_err_accuracy (d, b, _matrix_size);
      bench->accuracy->RMS = RMS (d, b, _matrix_size);
      bench->accuracy->forward_error = forward_error (d, b, _matrix_size);

      print_diff_accuracy ("Comparaison", buffer, bench, &data_2);

      free (a);
      free (b);
      free (c);
      free (d);
      save_data (filename, buffer);
    }
}

void
kernel1 (char *name_kernel, void (*kernel) (double *, double *, int),
         char *filename, struct bench *bench, int matrix_size)
{
  char buffer[1000];
  print_header_benchmark (buffer);
  for (int i = bench->start_size; i < bench->end_size; i += bench->step_size)
    {
      int _matrix_size = i;
      bench->data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;
      double *a, *b, *d_a, *d_b;
      a = (double *)malloc (sizeof (double) * _matrix_size_2);
      b = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (double));

      // copying init values
      for (int i = 0; i < _matrix_size; i++)
        {
          a[i] += drand48 ();
          b[i] = a[i];
        }
      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      DRIVER_BODY (kernel, d_a, d_b, _matrix_size);
      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      formatting_data (bench->data);
      print_data_benchmark (name_kernel, bench->data, buffer);
      save_data (filename, buffer);
    }
}

void
inversion (char *name_kernel, void (*kernel) (double *, double *, int),
           char *filename, struct bench *bench, int matrix_size)
{
  char buffer[1000];
  print_header_diff (buffer);
  for (int i = bench->start_size; i < bench->end_size; i += bench->step_size)
    {
      int _matrix_size = i;
      bench->data->matrice_size = _matrix_size;
      long _matrix_size_2 = _matrix_size * _matrix_size;

      double *a, *b, *c, *d, *d_a, *d_b, *d_c, *d_d;
      a = (double *)malloc (sizeof (double) * _matrix_size_2);
      b = (double *)malloc (sizeof (double) * _matrix_size_2);
      c = (double *)malloc (sizeof (double) * _matrix_size_2);
      d = (double *)malloc (sizeof (double) * _matrix_size_2);
      hipMalloc (&d_a, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_b, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_c, _matrix_size_2 * sizeof (double));
      hipMalloc (&d_d, _matrix_size_2 * sizeof (double));
      INIT (a, _matrix_size_2);
      INIT (b, _matrix_size_2);

      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      DRIVER_BODY (kernel, d_a, d_b, i);
      set_identity_matrix (c, i, i);
      hipMemcpy (d_a, a, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      hipMemcpy (d_c, c, _matrix_size_2 * sizeof (double),
                  hipMemcpyHostToDevice);
      ieee_64bits_gemm_jik (d_c, d_b, d_d, i);
      DRIVER_BODY (kernel, d_d, d_b, i);
      hipMemcpy (b, d_b, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      hipMemcpy (a, d_a, _matrix_size_2 * sizeof (double),
                  hipMemcpyDeviceToHost);
      formatting_data (bench->data);
      print_data_benchmark (name_kernel, bench->data, buffer);

      bench->accuracy->accuracy = compute_err_accuracy (a, b, i);
      bench->accuracy->RMS = RMS (a, b, i);
      bench->accuracy->forward_error = forward_error (a, b, i);

      print_diff_accuracy ("Comparaison", buffer, bench, bench->data);
      free (a);
      free (b);
      free (c);
      free (d);
      save_data (filename, buffer);
    }
}
/* API to run benchmark
   need to choose the accuracy driver */
void
benchmark (char *name_kernel, char *name_kernel_2, char *filename,
           void (*kernel) (double *, double *, int),
           void (*kernel_2) (double *, double *, int), struct bench *bench,
           enum benchmark_type type, int matrix_size)
{
  switch (type)
    {
    case INVERSION:
      inversion (name_kernel, kernel, filename, bench, matrix_size);
      break;
    case KERNEL1:
      kernel1 (name_kernel, kernel, filename, bench, matrix_size);
      break;
    case KERNEL2:
      kernel2 (name_kernel, name_kernel_2, kernel, kernel_2, filename, bench,
               matrix_size);
      break;
    }
}
