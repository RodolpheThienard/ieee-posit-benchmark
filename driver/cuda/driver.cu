  

extern "C"
{
#include "../../include/utils.h"
}
/* TODO Driver macro bandwidth
   macro permettant de faire la mesure du temps / débit
   paramettre variable
   Meta répétition de 31 pour stabilité avec 31 fois le meme calcul exact
   */
#define DRIVER_BANDWIDTH(fn, ...)                              \
  {                                                                           \
    dim3 threadsPerBlock (64, 64);                                            \
    dim3 numBlocks (size / threadsPerBlock.x,                          \
                    size / threadsPerBlock.y);                         \
    hipEvent_t start_event, end_event;                                       \
    float elapsed;                                                            \
    for (int stability = 0; stability < 33; stability++)                      \
      {                                                                       \
        hipEventCreate (&start_event);                                       \
        hipEventCreate (&end_event);                                         \
        hipEventRecord (start_event, 0);                                     \
        for (int rep = 0; rep < bench->data->repetition; rep++)               \
          {                                                                   \
            fn<<<threadsPerBlock, numBlocks>>> (__VA_ARGS__);                 \
          }                                                                   \
        hipEventRecord (end_event, 0);                                       \
        hipEventSynchronize (end_event);                                     \
        hipEventElapsedTime (&elapsed, start_event, end_event);              \
        elapsed *= 1e-3;                                                      \
        bench->data->samples[stability]                                       \
            = (double)elapsed / bench->data->repetition;                      \
      }                                                                       \
  }

/* TODO driver macro accuracy
   macro permettant de faire la mesure de précision
   verification de la sorti du calcul de Bandwidth avec la fonction CPU
   Checker si OMP et x86 sont équivalent */
#define DRIVER_ACCURACY(size, host, device, bench)                            \
  bench->accuracy->accuracy = compute_err_accuracy_float (host, device, size);      \
  bench->accuracy->RMS = RMS_float (host, device, size);                            \
  bench->accuracy->forward_error = forward_error_float (host, device, size);

/* Foo example of API utilisation
   compare 2 function for dgemm */
void
driver_sgemm (void (*function) (float *, float *, float *, int), int size,
              float *a, float *b, float *c, struct bench_s bench[])
{
  DRIVER_BANDWIDTH (function, a, b, c, size);
  formatting_data (bench->data);
}

void
driver_accuracy_32bits (int size, float *c_host, float *c_device,
                        struct bench_s bench[])
{
  DRIVER_ACCURACY ( size, c_host, c_device, bench);
}
