extern "C"
{
#include "../../include/utils.h"
}

/* TODO Driver macro bandwidth
   macro permettant de faire la mesure du temps / débit
   paramettre variable
   Meta répétition de 31 pour stabilité avec 31 fois le meme calcul exact
   */
#define DRIVER_BANDWIDTH(fn, ...)                              \
  {                                                                           \
    dim3 threadsPerBlock (64, 64);                                            \
    dim3 numBlocks (size / threadsPerBlock.x,                          \
                    size / threadsPerBlock.y);                         \
    hipEvent_t start_event, end_event;                                       \
    float elapsed;                                                            \
    for (int stability = 0; stability < 33; stability++)                      \
      {                                                                       \
        hipEventCreate (&start_event);                                       \
        hipEventCreate (&end_event);                                         \
        hipEventRecord (start_event, 0);                                     \
        for (int rep = 0; rep < bench->data->repetition; rep++)               \
          {                                                                   \
            fn<<<threadsPerBlock, numBlocks>>> (__VA_ARGS__);                 \
          }                                                                   \
        hipEventRecord (end_event, 0);                                       \
        hipEventSynchronize (end_event);                                     \
        hipEventElapsedTime (&elapsed, start_event, end_event);              \
        elapsed *= 1e-3;                                                      \
        bench->data->samples[stability]                                       \
            = (double)elapsed / bench->data->repetition;                      \
      }                                                                       \
  }

#define DRIVER_ACCURACY(fn, ...)                                              \
  {                                                                           \
    dim3 threadsPerBlock (64, 64);                                            \
    dim3 numBlocks (size / threadsPerBlock.x,                          \
                    size / threadsPerBlock.y);                         \
    fn<<<threadsPerBlock, numBlocks>>> (__VA_ARGS__);                         \
  }
  
/* TODO driver macro accuracy
   macro permettant de faire la mesure de précision
   verification de la sorti du calcul de Bandwidth avec la fonction CPU
   Checker si OMP et x86 sont équivalent */

#define DRIVER_ACCURACY_COMPARE(size, host, device, bench)                         \
  bench->accuracy->accuracy = compute_err_accuracy (host, device, size);      \
  bench->accuracy->RMS = RMS (host, device, size);                            \
  bench->accuracy->forward_error = forward_error (host, device, size);


void
driver_accuracy (int size, double *c_host, double *c_device,
                        struct bench_s bench[])
{
  DRIVER_ACCURACY_COMPARE ( size, c_host, c_device, bench);
}

/* Foo example of API utilisation
   compare 2 function for dgemm */
void
driver_sgemm (void (*function) (float *, float *, float *, int), int size,
              float *a, float *b, float *c, struct bench_s bench[])
{
  DRIVER_BANDWIDTH (function, a, b, c, size);
  hipMemset(c,0,size*size*sizeof(float));
  DRIVER_ACCURACY(function, a,b,c,size);
  formatting_data (bench->data);
}

void
driver_inverse_gauss_jordan (void (*function) (float *, float *, int, int), int size,
              float *a, float *b, int i, struct bench_s bench[])
{
  DRIVER_BANDWIDTH(function, a,b,size, i);
  hipMemset(b,0,size*size*sizeof(float));
  DRIVER_ACCURACY(function, a,b,size, i);
  formatting_data (bench->data);
}
