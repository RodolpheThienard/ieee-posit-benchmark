  

extern "C"
{
#include "../../include/utils.h"
}
/* TODO Driver macro bandwidth
   macro permettant de faire la mesure du temps / débit
   paramettre variable
   Meta répétition de 31 pour stabilité avec 31 fois le meme calcul exact
   */
#define DRIVER_BANDWIDTH(fn, ...)                              \
  {                                                                           \
    dim3 threadsPerBlock (32, 32);                                            \
    dim3 numBlocks (size / threadsPerBlock.x,                          \
                    size / threadsPerBlock.y);                         \
    hipEvent_t start_event, end_event;                                       \
    float elapsed;                                                            \
    for (int stability = 0; stability < 33; stability++)                      \
      {                                                                       \
        hipEventCreate (&start_event);                                       \
        hipEventCreate (&end_event);                                         \
        hipEventRecord (start_event, 0);                                     \
        for (int rep = 0; rep < bench->data->repetition; rep++)               \
          {                                                                   \
            fn<<<32,32>>> (__VA_ARGS__);                 \
          }                                                                   \
        hipEventRecord (end_event, 0);                                       \
        hipEventSynchronize (end_event);                                     \
        hipEventElapsedTime (&elapsed, start_event, end_event);              \
        elapsed *= 1e-3;                                                      \
        bench->data->samples[stability]                                       \
            = (double)elapsed / bench->data->repetition;                      \
      }                                                                       \
  }

/* TODO driver macro accuracy
   macro permettant de faire la mesure de précision
   verification de la sorti du calcul de Bandwidth avec la fonction CPU
   Checker si OMP et x86 sont équivalent */
#define DRIVER_ACCURACY(host, device, size)                                   \
  bench->accuracy->accuracy = compute_err_accuracy (host, device, size);      \
  bench->accuracy->RMS = RMS (host, device, size);                            \
  bench->accuracy->forward_error = forward_error (host, device, size);

/* Foo example of API utilisation
   compare 2 function for dgemm */
void
driver_sgemm (void (*function) (float *, float *, float *, int), int size,
              float *a, float *b, float *c, struct bench_s bench[])
{
  DRIVER_BANDWIDTH (function, a, b, c, size);
  formatting_data (bench->data);
}

void
driver_accuracy_32bits (int size, float *c_host, float *c_device,
                        struct bench_s bench[])
{
  DRIVER_ACCURACY ((double *)c_host, (double *)c_device, size);
}
