extern "C" {
  #include "../include/driver.h"
}
  #include "../include/hip/hip_runtime.h"

#define DRIVER_BODY_BENCHMARK_CUDA(kernel, ...)                                         \
  dim3 threadsPerBlock (32, 32);                                              \
  dim3 numBlocks (matrix_size / threadsPerBlock.x,                            \
                  matrix_size / threadsPerBlock.y);                           \
  hipEvent_t start_event, end_event;                                         \
  float elapsed;                                                              \
  for (uint32_t stability = 0; stability < 33; stability++)                   \
    {                                                                         \
      hipEventCreate (&start_event);                                         \
      hipEventCreate (&end_event);                                           \
      hipEventRecord (start_event, 0);                                       \
      for (uint32_t rep = 0; rep < data->repetition; rep++)                   \
        {                                                                     \
          kernel<<<numBlocks, threadsPerBlock>>> (__VA_ARGS__);               \
        }                                                                     \
      hipEventRecord (end_event, 0);                                         \
      hipEventSynchronize (end_event);                                       \
      hipEventElapsedTime (&elapsed, start_event, end_event);                \
      elapsed *= 1e-3;                                                        \
      data->samples[stability] = (double)elapsed / data->repetition;          \
    }                                                                         

void
driver_cuda_fp64_benchmark (char *title, char *buffer,
                            void (*kernel) (double *, double *, double *, int),
                            struct data *data, int matrix_size)
{
  double *A, *B, *C, *d_A, *d_B, *d_C;
  long _matrix_size_2 = matrix_size * matrix_size;
  A = (double *)malloc (sizeof (double) * _matrix_size_2);
  B = (double *)malloc (sizeof (double) * _matrix_size_2);
  C = (double *)malloc (sizeof (double) * _matrix_size_2);

  INIT (A, _matrix_size_2);
  INIT (B, _matrix_size_2);
  INIT (C, _matrix_size_2);

  hipblasHandle_t handle;
  hipblasCreate (&handle);
  hipMalloc (&d_A, _matrix_size_2 * sizeof (double));
  hipMalloc (&d_B, _matrix_size_2 * sizeof (double));
  hipMalloc (&d_C, _matrix_size_2 * sizeof (double));

  hipMemcpy (d_A, A, _matrix_size_2 * sizeof (double),
              hipMemcpyHostToDevice);
  hipMemcpy (d_B, B, _matrix_size_2 * sizeof (double),
              hipMemcpyHostToDevice);

  DRIVER_BODY_BENCHMARK_CUDA (kernel, d_A, d_B, d_C, matrix_size);
  formatting_data (data);
  print_data_benchmark (title, data, buffer);

  hipFree (d_A);
  hipFree (d_B);
  hipFree (d_C);
  free (A);
  free (B);
  free (C);
}

